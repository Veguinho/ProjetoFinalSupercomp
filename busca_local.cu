#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <functional>
#include <fstream>
#include <algorithm>
#include <sstream>
#include <string>
#include <random>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>
using namespace std;

int calc_satisfacao_atual(vector<vector<int>>& prefs, vector<int>& aluno_projeto){
    int satisfacao_atual = 0;
    for(auto aluno_atual = 0; aluno_atual < aluno_projeto.size(); aluno_atual++){
        satisfacao_atual += prefs[aluno_atual][aluno_projeto[aluno_atual]];
    }
    return satisfacao_atual;
}

struct escolhe_alunos{
    int *prefs;
    int *aluno_projeto;
    int n_alunos;
    int n_projetos;
    int seed;
    thrust::uniform_int_distribution<int> dist;
    
    escolhe_alunos (int *prefs, int *aluno_projeto, int n_alunos, int n_projetos, int seed, 
                    thrust::uniform_int_distribution<int> dist) : prefs(prefs), 
                                                                  aluno_projeto(aluno_projeto),
                                                                  n_alunos(n_alunos),
                                                                  n_projetos(n_projetos),
                                                                  seed(seed),
                                                                  dist(dist){};

    __device__ __host__
    int operator()(const int &i) {;

    int random_int;
    thrust::default_random_engine rng(i+seed);
    rng.discard(i);

    int tmp1;
    for(int k=0; k < n_alunos; k++){
        random_int = dist(rng);
        tmp1 = aluno_projeto[k+i*n_alunos];
        aluno_projeto[k+i*n_alunos] = aluno_projeto[random_int+i*n_alunos]; //faz trocas no aluno projeto aleatoriamente
        aluno_projeto[random_int+i*n_alunos] = tmp1;
    }
    
    int satisfacao_local = 0;
    bool flag = true;

    //calcula satisfacao
    for(int c=0; c < n_alunos; c++){
        satisfacao_local += prefs[c*n_projetos+aluno_projeto[c+i*n_alunos]];
    }
    
    while(flag){
        flag = false;
        for(int a=0; a < n_alunos; a++){
            for(int b=0; b < n_alunos; b++){
                if(b>a){
                    int satisfacao_tmp = 0;
                    tmp1 = aluno_projeto[a+i*n_alunos];
                    aluno_projeto[a+i*n_alunos] = aluno_projeto[b+i*n_alunos]; //faz trocas no aluno projeto
                    aluno_projeto[b+i*n_alunos] = tmp1;

                    //calcula satisfacao
                    for(int c=0; c < n_alunos; c++){
                        satisfacao_tmp += prefs[c*n_projetos+aluno_projeto[c+i*n_alunos]];
                    }

                    if(satisfacao_tmp > satisfacao_local){
                        satisfacao_local = satisfacao_tmp;
                        flag = true;
                    }
                    else{
                        tmp1 = aluno_projeto[a+i*n_alunos];
                        aluno_projeto[a+i*n_alunos] = aluno_projeto[b+i*n_alunos]; //destroca no aluno projeto
                        aluno_projeto[b+i*n_alunos] = tmp1;
                    }
                }
            }
        }
    }
    return satisfacao_local;
  }  
};

int main(){
    string line;
	vector<int> v_entrada;
    int tmp_op;
    getline(cin, line);
    istringstream ss(line);

    while(ss >> tmp_op){
        v_entrada.push_back(tmp_op);
    }
    int n_alunos, n_projetos, n_choices;
    n_alunos = v_entrada[0];
    n_projetos =  v_entrada[1];
    n_choices = v_entrada[2];

    vector<int> prefs(n_alunos*n_projetos);
    vector<int> projs;
    int proj_tmp;
 
    int seed = 0;
    int iterations = 100000;
    if(getenv("SEED")){
        seed = atoi(getenv("SEED"));
    }
    if(getenv("ITER")){
        iterations = atoi(getenv("ITER"));
    }

    for (int i=0; i<n_alunos; i++){       
        getline(cin, line);
        istringstream ss(line);
        projs.clear();
        while(ss >> proj_tmp){
            projs.push_back(proj_tmp);
        }
        for(int j = 0; j < n_choices; j++){     
            prefs[n_projetos*i+projs[j]] = pow(n_choices-j,2);
        }
    }
    
    vector<int> aluno_projeto_vector;
    //preenche aluno_projeto em ordem 0,0,0,1,1,1,2,2,2...
    for(int i=0; i<iterations; i++){
        vector<int> vagas(n_projetos,3);
        for (int proj = 0; proj < vagas.size(); proj++){
            while(vagas[proj]>0){
                aluno_projeto_vector.push_back(proj);
                vagas[proj] -=1;
            }
        }
    }

    thrust::uniform_int_distribution<int> dist(0,n_alunos-1);
    //Fazer um vetor gigante com varios vetores aleatorios contidos dentro dele
    //Passar esse vetor gigante no lugar deste "dist" abaixo
    thrust::host_vector<int> aluno_projeto_cpu(aluno_projeto_vector);
    thrust::host_vector<int> satisfacao_atual_cpu(iterations,-1);

    thrust::device_vector<int> satisfacao_atual_gpu(satisfacao_atual_cpu);
    thrust::device_vector<int> aluno_projeto_gpu(aluno_projeto_cpu);
    thrust::device_vector<int> prefs_gpu(prefs);

    escolhe_alunos calc_satisfac(thrust::raw_pointer_cast(prefs_gpu.data()),
                                thrust::raw_pointer_cast(aluno_projeto_gpu.data()),
                                n_alunos,
                                n_projetos,
                                seed,
                                dist);

    thrust::counting_iterator<int> iterator(0);
    thrust::transform(iterator, iterator+(iterations), satisfacao_atual_gpu.begin(), calc_satisfac);

    int index = 0;
    int best_index = 0;
    int best_result = 0;
    for (auto m = satisfacao_atual_gpu.begin(); m != satisfacao_atual_gpu.end(); m++){
        if(*m > best_result){
            best_result = *m;
            best_index = index;
        }
        index++;
    }

    cout << best_result << " 0\n";
    for (int m = best_index*n_alunos; m < (best_index+1)*n_alunos; m++){
        cout << aluno_projeto_gpu[m] << " ";
    }
    cout << "\n";
}